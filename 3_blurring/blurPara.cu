#include "hip/hip_runtime.h"
/*
=============================================================================================
N.R Cejas Bolecek ICNPG2017, June 5th 2017
#Adapted from udacity CS344 course: Intro to Parallel Programming
=============================================================================================
//Description:

// For a color image that has multiple channels, we suggest separating
// the different color channels so that each color is stored contiguously
// instead of being interleaved. This will simplify your code.

// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

//****************************************************************************

// The gaussian_blur kernel to perform the blurring of the
// inputChannel, using the array of weights, and put the result in the outputChannel.

// Here is an example of computing a blur, using a weighted average, for a single
// pixel in a small image.
//
// Array of weights:
//
//  0.0  0.2  0.0
//  0.2  0.2  0.2
//  0.0  0.2  0.0
//
// Image (note that we align the array of weights to the center of the box):
//
//    1  2  5  2  0  3
//       -------
//    3 |2  5  1| 6  0       0.0*2 + 0.2*5 + 0.0*1 +
//      |       |
//    4 |3  6  2| 1  4   ->  0.2*3 + 0.2*6 + 0.2*2 +   ->  3.2
//      |       |
//    0 |4  0  3| 4  2       0.0*4 + 0.2*0 + 0.0*3
//       -------
//    9  6  5  0  3  9
//
//         (1)                         (2)                 (3)
//
// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.
*/
#include "utils.h"
#include <stdio.h>

// GLOBAL VARS ON DEVICE!
unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

// devicel functions:
__device__ int min_int(int a, int b){
  if (a <= b) return a;
  else return b;
}
__device__ int max_int(int a, int b) {
  if (a>=b) return a;
  else return b;
}

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

  const int2 thread_2D_pos = make_int2(blockIdx.x*blockDim.x + threadIdx.x,
                                        blockIdx.y*blockDim.y + threadIdx.y);
  if(thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) return;
  const int thread_1D_pos = thread_2D_pos.y*numCols + thread_2D_pos.x;


  float result = 0.f;
  //For every value in the filter around the pixel (c, r)
  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {

      int image_r = min_int(max_int(thread_2D_pos.y + filter_r, 0), static_cast<int>(numRows - 1));
      int image_c = min_int(max_int(thread_2D_pos.x + filter_c, 0), static_cast<int>(numCols - 1));

      if (thread_1D_pos == 0) {
        printf("filter_r: %d, filter_c: %d, image_r: %d, image_c: %d \n",filter_r,filter_c,image_r,image_c);
      }

      float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
      float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

      result += image_value * filter_value;
    }
  }

  outputChannel[thread_1D_pos] = result;

}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{


  const int2 thread_2D_pos = make_int2(blockIdx.x*blockDim.x + threadIdx.x,
                                       blockIdx.y*blockDim.y + threadIdx.y);

  if(thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) return;

  const int thread_1D_pos = thread_2D_pos.y*numCols + thread_2D_pos.x;

  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;

}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;


  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}


void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  // Now allocate memory for filter and copy from host to device ptr.
  checkCudaErrors(hipMalloc((void**)&d_filter,sizeof(float)*filterWidth*filterWidth));
  checkCudaErrors(hipMemcpy(d_filter,h_filter,sizeof(float)*filterWidth*filterWidth,
                             hipMemcpyHostToDevice));

  // Testing - inspect h_filter:
  //for(int i=0; i<filterWidth; i++) {
  //  for(int j=0; j<filterWidth; j++) {
  //    printf("i: %d, j: %d, val: %f \n",i,j,*(h_filter +i*filterWidth + j));
  //  }
  //}

}

void blurPara(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred,
                        unsigned char *d_greenBlurred,
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{

  // printf("numRows: %lu, numCols: %lu\n",numRows,numCols);

  const dim3 blockSize(20,20,1);  // 20*20 = 400 threads per block

  const dim3 gridSize(numCols/blockSize.x+1,numRows/blockSize.y+1,1);

  separateChannels<<<gridSize,blockSize>>>(d_inputImageRGBA,numRows,numCols,
                                           d_red,d_green,d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  gaussian_blur<<<gridSize,blockSize>>>(d_red,d_redBlurred,numRows,numCols,
                                        d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize,blockSize>>>(d_green,d_greenBlurred,numRows,numCols,
                                        d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize,blockSize>>>(d_blue,d_blueBlurred,numRows,numCols,
                                        d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}

//Free all the memory that we allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
